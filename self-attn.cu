#include "hip/hip_runtime.h"
#include "zksoftmax.cuh"
#include "zkfc.cuh"
#include "fr-tensor.cuh"
#include "proof.cuh"
#include "commitment.cuh"
#include "rescaling.cuh"
#include <string>

FrTensor rotate_half(const FrTensor& X, uint seq_len, uint num_head, uint head_dim)
{
    auto X_T = X.transpose(seq_len * num_head, head_dim);
    auto x1 = X_T.trunc(0, X_T.size >> 1);
    auto x2 = X_T.trunc(X_T.size >> 1, X_T.size);
    return catTensors({-x2, x1}).transpose(head_dim, seq_len * num_head);
}

int main(int argc, char *argv[])
{
    string input_file_name = argv[1];
    uint seq_len = std::stoi(argv[2]);
    uint num_head = std::stoi(argv[3]);
    uint head_dim = std::stoi(argv[4]);
    uint embed_dim = num_head * head_dim;
    string workdir = argv[5];
    string layer_prefix = argv[6];
    string output_file_name = argv[7];

    auto q_proj = create_weight(
        workdir + "/self_attn.q_proj.weight-pp.bin",
        workdir + "/" + layer_prefix + "-self_attn.q_proj.weight-int.bin",
        workdir + "/" + layer_prefix + "-self_attn.q_proj.weight-commitment.bin",
        embed_dim,
        embed_dim
    );

    auto k_proj = create_weight(
        workdir + "/self_attn.k_proj.weight-pp.bin",
        workdir + "/" + layer_prefix + "-self_attn.k_proj.weight-int.bin",
        workdir + "/" + layer_prefix + "-self_attn.k_proj.weight-commitment.bin",
        embed_dim,
        embed_dim
    );

    auto v_proj = create_weight(
        workdir + "/self_attn.v_proj.weight-pp.bin",
        workdir + "/" + layer_prefix + "-self_attn.v_proj.weight-int.bin",
        workdir + "/" + layer_prefix + "-self_attn.v_proj.weight-commitment.bin",
        embed_dim,
        embed_dim
    );

    auto o_proj = create_weight(
        workdir + "/self_attn.o_proj.weight-pp.bin",
        workdir + "/" + layer_prefix + "-self_attn.o_proj.weight-int.bin",
        workdir + "/" + layer_prefix + "-self_attn.o_proj.weight-commitment.bin",
        embed_dim,
        embed_dim
    );

    zkFC q_layer(embed_dim, embed_dim, q_proj.weight);
    zkFC k_layer(embed_dim, embed_dim, k_proj.weight);
    zkFC v_layer(embed_dim, embed_dim, v_proj.weight);
    zkFC o_layer(embed_dim, embed_dim, o_proj.weight);

    auto cos = FrTensor::from_int_bin("cos_temp.bin");
    auto sin = FrTensor::from_int_bin("sin_temp.bin");

    Rescaling q_rescale(1 << 16);
    Rescaling q_new_rescale(1 << 16);
    Rescaling k_rescale(1 << 16);
    Rescaling k_new_rescale(1 << 16);
    Rescaling v_rescale(1 << 16);
    Rescaling o_rescale(1 << 16);
    Rescaling y_rescale(1 << 16);

    FrTensor input = FrTensor::from_int_bin(input_file_name);
    // cout << input.size << endl;
    auto Q = q_layer(input);
    auto Q_ = q_rescale(Q);
    
    auto Q_rotated = rotate_half(Q_, seq_len, num_head, head_dim);
    auto Q_new = Q_ * cos + Q_rotated * sin;
    auto Q_new_ = q_new_rescale(Q_new);

    auto K = k_layer(input);
    auto K_ = k_rescale(K);
    
    auto K_rotated = rotate_half(K_, seq_len, num_head, head_dim);
    auto K_new = K_ * cos + K_rotated * sin;
    auto K_new_ = k_new_rescale(K_new);

    auto V = v_layer(input);
    auto V_ = v_rescale(V);
    

    zkAttn attn(1L << 16, 1L << 16, {1 << 16, 1 << 16, 1 << 16}, 1, 0, {1.0 * (1L << 5), 1.0 * (1L << 11)}, seq_len, seq_len, embed_dim, 1 << 12);

    // CACHES
    FrTensor sm_in(seq_len * seq_len), sm_out(seq_len * seq_len), sm_shift(seq_len), sm_in_shifted(seq_len * seq_len);
    vector<FrTensor> sm_in_segments, sm_out_segments, sm_m_segments;

    auto Y = attn.compute(Q_new_, K_new_, V_, sm_in, sm_out, sm_shift, sm_in_shifted, sm_in_segments, sm_out_segments, sm_m_segments);
    
    auto Y_ = y_rescale(Y);
    auto O = o_layer(Y_);
    auto O_ = o_rescale(O);
    

    O_.save_int(output_file_name);

    o_rescale.prove(O, O_);

    y_rescale.prove(Y, Y_);

    attn.prove(Q_new_, K_new_, V_, Y, sm_out, sm_in, sm_shift, sm_in_shifted, sm_in_segments, sm_out_segments, sm_m_segments);

    q_new_rescale.prove(Q_new, Q_new_);
    q_rescale.prove(Q, Q_);
    hadamard_product_sumcheck(Q_, cos, random_vec(ceilLog2(Q_.size)), random_vec(ceilLog2(Q_.size)));
    hadamard_product_sumcheck(Q_rotated, sin, random_vec(ceilLog2(Q_rotated.size)), random_vec(ceilLog2(Q_rotated.size)));
    k_new_rescale.prove(K_new, K_new_);
    k_rescale.prove(K, K_);
    hadamard_product_sumcheck(K_, cos, random_vec(ceilLog2(K_.size)), random_vec(ceilLog2(K_.size)));
    hadamard_product_sumcheck(K_rotated, sin, random_vec(ceilLog2(K_rotated.size)), random_vec(ceilLog2(K_rotated.size)));
    v_rescale.prove(V, V_);

    verifyWeightClaim(o_proj, o_layer.prove(Y_, O)[0]);
    verifyWeightClaim(k_proj, k_layer.prove(input, K)[0]);
    verifyWeightClaim(q_proj, q_layer.prove(input, Q)[0]);
    verifyWeightClaim(v_proj, v_layer.prove(input, V)[0]);

    // cout << O_(0) << " " << O_(O_.size - 1) << endl;
    return 0;
}